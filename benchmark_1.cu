#include "hip/hip_runtime.h"

#define GOOD_WEATHER 0
#define BAD_WEATHER 1

#define TAG_Car 0
#define TAG_Pedestrian 1

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <random>
#include <array> 
#include <algorithm> 

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUM_CARS 1024
#define NUM_PEDS 4096
#define NUM_STREETS 500
#define MAX_CONNECTIONS 5
#define MAX_LEN 25

using namespace std;

__device__ float *Actor_progress;
__device__ int *Actor_street;
__device__ float *Car_max_velocity;
__device__ float *Street_length;
__device__ float *Street_max_velocity;
__device__ int *Street_neighbors;
__device__ int *Array_Street_size;
__device__ int *Array_Street_offset;
__device__ int *Array_Street_arrays;
__device__ int *input_actor_tag;
__device__ int *input_actor_id;
__device__ int *jobs;
__shared__ hiprandState_t rand_state;

__device__ void method_Car_move(int actor_id, int weather)
{
	float weather_multiplier;
	if (weather == GOOD_WEATHER) 
	{
		weather_multiplier = 1.0
	}
	else if (weather == BAD_WEATHER)
	{
		weather_multiplier = 0.75;
	}

	float speed = min(Car_max_velocity[actor_id], Street_max_velocity[Actor_street[actor_id]]) * weather_multiplier;
	Actor_progress[actor_id] = Actor_progress[actor_id] + (speed / 60.0); /* 1 tick = 1 minute */

	if (Actor_progress[actor_id] >= Street_length[Actor_street[actor_id]])
	{
		// move to different street
		int array_id = Street_neighbors[Actor_street[actor_id]];
		int neighbor_index = hiprand(&state) % Array_Street_size[array_id];
		Actor_street[actor_id] = Array_Street_arrays[Array_Street_offset[array_id] + neighbor_index];
	}
}

__device__ void method_Pedestrian_move(int actor_id, int weather)
{
	float weather_multiplier;
	if (weather == GOOD_WEATHER) 
	{
		weather_multiplier = 1.0
	}
	else if (weather == BAD_WEATHER)
	{
		weather_multiplier = 0.75;
	}

	float speed = hiprand(&state) % 7 - 2;
	Actor_progress[actor_id] = Actor_progress[actor_id] + (speed / 60.0); /* 1 tick = 1 minute */

	if (Actor_progress[actor_id] >= Street_length[Actor_street[actor_id]])
	{
		// move to different street
		int array_id = Street_neighbors[Actor_street[actor_id]];
		int neighbor_index = hiprand(&state) % Array_Street_size[array_id];
		Actor_street[actor_id] = Array_Street_arrays[Array_Street_offset[array_id] + neighbor_index];
	}
}

__device__ void block(int actor_tag, int actor_id, int weather, int ticks)
{
	for (int i = 0; i < ticks; i++)
	{
		switch (actor_tag)
		{
			case TAG_Car:
				method_Car_move(actor_id, weather);
				break;
			case TAG_Pedestrian:
				method_Pedestrian_move(actor_id, weather);
				break;
		}
	}
}

__global__ void kernel(int weather,	int ticks)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (threadIdx.x == 1)
	{
		hiprand_init(42, 0, 0, &state);
	}

	__syncthreads();

	block(input_actor_tag[jobs[tid]], input_actor_id[jobs[tid]], weather, ticks);
}

int main()
{
	printf("Setting up scenario...\n");
	srand(42);

	// streets
	int *Street_length = new int[NUM_STREETS];
	int *Street_max_velocity = new int[NUM_STREETS];
	int *Street_neighbors = new int[NUM_STREETS];

	for (int i = 0; i < NUM_STREETS; i++)
	{
		Street_length[i] = rand() % MAX_LEN + 1;
		Street_max_velocity[i] = rand() % 40 + 45;	/* speed between 45 and 105 */
		Street_neighbors[i] = i;
	}

	// neighbors
	int *Array_Street_offset = new int[NUM_STREETS];
	int *Array_Street_size = new int[NUM_STREETS];
	int num_connections = 0;

	for (int i = 0; i < NUM_STREETS; i++)
	{
		Array_Street_offset[i] = num_connections;
		int connections = rand() % MAX_CONNECTIONS + 1
		Array_Street_size[i] = connections;
		num_connections += connections;
	}

	int *Array_Street_arrays = new int[num_connections];
	for (int i = 0; i < num_connections; i++)
	{
		Array_Street_arrays[i] = rand() % NUM_STREETS;
	}

	// actors
	int *Actor_street = new int[NUM_PEDS + NUM_CARS];
	float *Actor_progress = new int[NUM_PEDS + NUM_CARS];
	int *Car_max_velocity = new float[NUM_CARS + NUM_PEDS];
	int *Actor_tag = new int[NUM_PEDS + NUM_CARS];
	int *Actor_id = new int[NUM_PEDS + NUM_CARS];

	for (int i = 0; i < NUM_PEDS + NUM_CARS; i++)
	{
		Actor_street[i] = rand() % NUM_STREETS;
		Actor_progress[i] = 0.0f;
		Car_max_velocity[i] = rand() % 20 + 65;
	}

	for (int i = 0; i < NUM_PEDS; i++)
	{
		Actor_tag[i] = TAG_Pedestrian;
		Actor_id[i] = i;
	}

	for (int i = NUM_PEDS; i < NUM_PEDS + NUM_CARS; i++)
	{
		Actor_tag[i] = TAG_Car;
		Actor_id[i] = i;
	}

	shuffle(Actor_tag, Actor_tag + NUM_CARS + NUM_PEDS, std::default_random_engine(42));

	// jobs (dummy)
	int *jobs = new int[NUM_PEDS + NUM_CARS];

	for (int i = 0; i < NUM_CARS + NUM_PEDS; i++)
	{
		jobs[i] = i;
	}

	printf("Scenario set up.\n");

	printf("Copying data to GPU...\n");
	hipMemcpyToSymbol(HIP_SYMBOL("Actor_progress"), &Actor_progress[0], sizeof(float) * (NUM_PEDS + NUM_CARS), size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("Actor_street"), &Actor_street[0], sizeof(int) * (NUM_PEDS + NUM_CARS), size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("Car_max_velocity"), &Car_max_velocity[0], sizeof(float) * (NUM_PEDS + NUM_CARS), size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("Street_length"), &Street_length[0], sizeof(float) * NUM_STREETS, size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("Street_max_velocity"), &Street_max_velocity[0], sizeof(float) * NUM_STREETS, size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("Street_neighbors"), &Street_neighbors[0], sizeof(int) * NUM_STREETS, size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("Array_Street_size"), &Array_Street_size[0], sizeof(int) * NUM_STREETS, size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("Array_Street_offset"), &Array_Street_offset[0], sizeof(int) * NUM_STREETS, size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("Array_Street_arrays"), &Array_Street_arrays[0], sizeof(int) * num_connections, size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("input_actor_tag"), &Actor_tag[0], sizeof(int) * (NUM_PEDS + NUM_CARS), size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("input_actor_id"), &Actor_id[0], sizeof(int) * (NUM_PEDS + NUM_CARS), size_t(0), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL("jobs"), &jobs[0], sizeof(int) * (NUM_PEDS + NUM_CARS), size_t(0), hipMemcpyHostToDevice);
	printf("Finished copying data.\n");

	printf("Launching kernel...\n");
	kernel<<<dim3(32), dim3((NUM_PEDS + NUM_CARS) / 32)>>>(GOOD_WEATHER, 10);
	printf("Kernel finished.\n");
}